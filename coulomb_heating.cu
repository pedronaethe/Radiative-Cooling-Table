/*! \file cooling_wrapper.cu
 *  \brief Wrapper file for to load CUDA cooling tables. */



#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

hipTextureObject_t coulombTexObj;
hipArray *cuCoulombArray = 0;

// Load the cooling_table into the CPU Memory.
void Load_Coulomb_Tables(float *coulomb_table)
{
    double *ne_arr;
    double *te_arr;
    double *ti_arr;
    double *coulomb_arr;

    double ne;
    double te;
    double ti;
    double coulomb;

    int i = 0;
    int nx = 100; // Number of Te data.
    int ny = 100; // Number of ne data.
    int nz = 100; // Number of Bmag data.

    FILE *infile;

    // Allocate arrays for temperature, electronic density and radius data.
    ne_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    te_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    coulomb_arr = (double *)malloc(nx * ny *  nz * sizeof(double));
    ti_arr = (double *)malloc(nx * ny * nz * sizeof(double));

    // Reading the cooling table
    infile = fopen("source_coulomb.txt", "r"); // this command is to ignore the first line.

    if (infile == NULL)
    {
        printf("Unable to open cooling file.\n");
        exit(1);
    }

    fscanf(infile, "%*[^\n]\n");
    while (fscanf(infile, "%lf, %lf, %lf, %lf", &ne, &ti, &te, &coulomb) == 4)
    {
        ne_arr[i] = ne;
        te_arr[i] = te;
        ti_arr[i] = ti;
        coulomb_arr[i] = coulomb;

        i++;
    }

    fclose(infile);
    // copy data from cooling array into the table
    for (i = 0; i < nx * ny * nz; i++)
    {
        coulomb_table[i] = float(coulomb_arr[i]);
    }

    // Free arrays used to read in table data
    free(ne_arr);
    free(te_arr);    
    free(ti_arr);
    free(coulomb_arr);
}

void CreateTexture()
{

    float *coulomb_table; //Device Array with cooling floats
    // number of elements in each variable
    const int nx = 100; //te
    const int ny = 100; //ti
    const int nz = 100; //ne
    coulomb_table = (float *)malloc(nx* ny * nz * sizeof(float));
    Load_Coulomb_Tables(coulomb_table); //Loading Cooling Values into pointer
    //cudaArray Descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    //cuda Array
    hipArray *cuCoulombArray;
    //checkCudaErrors(cudaMalloc3DArray(&cuCoolArray, &channelDesc, make_cudaExtent(nx*sizeof(float),ny,nz), 0));
    hipMalloc3DArray(&cuCoulombArray, &channelDesc, make_hipExtent(nx,ny,nz), 0);
    hipMemcpy3DParms copyParams = {0};

    //Array creation
    copyParams.srcPtr   = make_hipPitchedPtr((void *) coulomb_table, nx*sizeof(float), nx, ny);
    copyParams.dstArray = cuCoulombArray;
    copyParams.extent   = make_hipExtent(nx,ny,nz);
    copyParams.kind     = hipMemcpyHostToDevice;
    //checkCudaErrors(cudaMemcpy3D(&copyParams));
    hipMemcpy3D(&copyParams);
    //Array creation End

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(texRes));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = cuCoulombArray;
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(texDescr));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    //checkCudaErrors(cudaCreateTextureObject(&coolTexObj, &texRes, &texDescr, NULL));}
    hipCreateTextureObject(&coulombTexObj, &texRes, &texDescr, NULL);
}
__global__ void coulomb_function(hipTextureObject_t my_tex, float a1, float a2, float a3)
{
    float v1, v2, v3, lambda;

    //Values for testing;
    v1 = a1; //ne parameter
    v2 = a2; //ti parameter
    v3 = a3; //te parameter
    printf("Values you chose:\n");
    printf("ne = %f, ti= %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //The remapping formula goes (variable - initial_value) * (N - 1)/(max_value - init_value)
    // const int nx = 70; //Number of te used to generate table
    // const int ny = 70; //Number of ne used to generate table
    // const int nz = 70; //Number of r used to generate table
    //v1 = round((v1 - 6) * (nz - 1)/6);
    //v2 = round((v2 - 12) * (ny - 1)/8);
    //v3 = round((v3 - 6) * (nx - 1)/4);
    //printf("a = %f, b = %f, c = %f\n", v1, v2, v3);

    // For the normalized version only.
    const int nx = 100; //Number of te used to generate table
    const int ny = 100; //Number of ti used to generate table
    const int nz = 100; //Number of ne used to generate table
     v1 = (round((v1 - 12) * (nz - 1)/10) + 0.5 )/nz;
     v2 = (round((v2 - 4) * (ny - 1)/11) + 0.5 )/ny;
     v3 = (round((v3 - 4) * (nx - 1)/11) + 0.5 )/nx;

    printf("Coordinates in texture grid:\n");
    printf("ne = %f, ti = %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //lambda = tex3D<float>(coolTexObj, v3 + 0.5f, v2 + 0.5f, v1 + 0.5f); 

    // //For the normalized version only.
    lambda = tex3D<float>(my_tex, v3, v2, v1); 
    printf("Cooling value = %lf\n", lambda);
    return;
}

int main()
{
    float read1, read2, read3;
    float loop = 100;
    char str[1];
    CreateTexture();
    while (loop > 1)
    {
	    printf("ne value:\n");
	    scanf("%f", &read1);
	    printf("ti value:\n");
	    scanf("%f", &read2);
	    printf("Te value:\n");
	    scanf("%f", &read3);
	    coulomb_function<<<1, 1>>>(coulombTexObj, read1, read2, read3);
        sleep(1);
	    printf("Do you want to read other values? y/n\n");
	    scanf("%s", str);
	    if (strcmp(str, "n") == 0)
	    {
	    	loop = 0;
	    }
	}
    hipDestroyTextureObject(coulombTexObj);
    return 0;
}
//DEPRECATED Texture Reference in CUDA 11.0
/*
//Texture and cudaArray declaration.
 texture<float, 3, cudaReadModeElementType> coolTexObj;
cudaArray *cuCoolArray = 0;


// Load the cooling_table into the CPU Memory.
void Load_Cooling_Tables(float *cooling_table)
{
    double *ne_arr;
    double *te_arr;
    double *bmag_arr;
    double *cool_arr;

    double ne;
    double te;
    double bmag;
    double cool;

    int i = 0;
    int nx = 100; // Number of Te data.
    int ny = 100; // Number of ne data.
    int nz = 100; // Number of Bmag data.

    FILE *infile;

    // Allocate arrays for temperature, electronic density and radius data.
    ne_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    te_arr = (double *)malloc(nx * ny * nz * sizeof(double));
    cool_arr = (double *)malloc(nx * ny *  nz * sizeof(double));
    bmag_arr = (double *)malloc(nx * ny * nz * sizeof(double));

    // Reading the cooling table
    infile = fopen("cooling_table_log_mag.txt", "r"); // this command is to ignore the first line.

    if (infile == NULL)
    {
        printf("Unable to open cooling file.\n");
        exit(1);
    }

    fscanf(infile, "%*[^\n]\n");
    while (fscanf(infile, "%lf, %lf, %lf, %lf", &bmag, &ne, &te, &cool) == 4)
    {
        ne_arr[i] = ne;
        te_arr[i] = te;
        bmag_arr[i] = bmag;
        cool_arr[i] = cool;

        i++;
    }

    fclose(infile);
    // copy data from cooling array into the table
    for (i = 0; i < nx * ny * nz; i++)
    {
        cooling_table[i] = float(cool_arr[i]);
    }

    // Free arrays used to read in table data
    free(ne_arr);
    free(te_arr);    
    free(bmag_arr);
    free(cool_arr);
}

 // \brief Load the Cloudy cooling tables into texture memory on the GPU. 
void Load_Cuda_Textures()
{

    float *cooling_table;

    // number of elements in each variable
    const int nx = 100; //te
    const int ny = 100; //ne
    const int nz = 100; //bmag


    // allocate host arrays to be copied to textures
    cooling_table = (float *)malloc(nx* ny * nz * sizeof(float));

    // Load cooling tables into the host arrays
    Load_Cooling_Tables(cooling_table);

    // Allocate CUDA arrays in device memory
    // The value of 64 in the CUDA channel must be checked, otherwise use 32 for float.
    cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
    cudaExtent volumeSize = make_cudaExtent(nx, ny, nz);
    cudaMalloc3DArray(&cuCoolArray, &channelDesc, volumeSize);

    // Copy to device memory the cooling and heating arrays
    // in host memory
    cudaMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_cudaPitchedPtr((void *)cooling_table, nx * sizeof(float), nx, ny); 
    copyParams.dstArray = cuCoolArray;
    copyParams.extent = volumeSize;
    copyParams.kind = cudaMemcpyHostToDevice;
    cudaMemcpy3D(&copyParams);

    // Specify texture reference parameters (same for both tables)
    coolTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.addressMode[2] = cudaAddressModeClamp; // out-of-bounds fetches return border values
    coolTexObj.filterMode = cudaFilterModeLinear;     // bi-linear interpolation
    coolTexObj.normalized = true;                     // Normalization of logarithm scale going from 0 to 1

    // Command to bind the array into the texture
    cudaBindTextureToArray(coolTexObj, cuCoolArray);
    // Free the memory associated with the cooling tables on the host
    free(cooling_table);
}

void Free_Cuda_Textures()
{
    // unbind the cuda textures
    cudaUnbindTexture(coolTexObj);
    // Free the device memory associated with the cuda arrays
    cudaFreeArray(cuCoolArray);
}

//Function used to interpolate the values of the cooling table.
__global__ void cooling_function(float a1, float a2, float a3)
{
    float v1, v2, v3, lambda;

    //Values for testing;
    v1 = a1; //Bmag parameter
    v2 = a2; //ne parameter
    v3 = a3; //te parameter
    printf("Values you chose:\n");
    printf("Bmag = %f, ne = %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //The remapping formula goes (variable - initial_value) * (N - 1)/(max_value - init_value)
    // const int nx = 70; //Number of te used to generate table
    // const int ny = 70; //Number of ne used to generate table
    // const int nz = 70; //Number of r used to generate table
    //v1 = round((v1 - 6) * (nz - 1)/6);
    //v2 = round((v2 - 12) * (ny - 1)/8);
    //v3 = round((v3 - 6) * (nx - 1)/4);
    //printf("a = %f, b = %f, c = %f\n", v1, v2, v3);

    // For the normalized version only.
    const int nx = 100; //Number of te used to generate table
    const int ny = 100; //Number of ne used to generate table
    const int nz = 100; //Number of Bmag used to generate table
     v1 = (round((v1 - 0.1) * (nz - 1)/9.99) + 0.5)/nz;
     v2 = (round((v2 - 12) * (ny - 1)/10) + 0.5 )/ny;
     v3 = (round((v3 - 4) * (nx - 1)/11) + 0.5 )/nx;

    printf("Coordinates in texture grid:\n");
    printf("Bmag = %f, ne = %f, Te = %f\n", v1, v2, v3);

    //For the non normalized version only.
    //lambda = tex3D<float>(coolTexObj, v3 + 0.5f, v2 + 0.5f, v1 + 0.5f); 

    // //For the normalized version only.
    lambda = tex3D<float>(coolTexObj, v3, v2, v1); 
    printf("Cooling value = %lf\n", lambda);
    return;
}

int main()
{
    float read1, read2, read3;
    float loop = 100;
    char str[1];
    Load_Cuda_Textures();
    while (loop > 1)
    {
	    printf("Bmag value:\n");
	    scanf("%f", &read1);
	    printf("ne value:\n");
	    scanf("%f", &read2);
	    printf("Te value:\n");
	    scanf("%f", &read3);
	    cooling_function<<<1, 1>>>(read1, read2, read3);
        sleep(1);
	    printf("Do you want to read other values? y/n\n");
	    scanf("%s", str);
	    if (strcmp(str, "n") == 0)
	    {
	    	loop = 0;
	    }
	}
    Free_Cuda_Textures();

    return 0;
}*/